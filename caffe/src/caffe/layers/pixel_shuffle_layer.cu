#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layers/pixel_shuffle_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void PixelShuffleForward(const int nthreads, const Dtype* bottom_data, Dtype* top_data, int sigma,
    int bottom_h, int bottom_w, int bottom_c, int shuffled_channels, int shuffled_height, int shuffled_width) {
  CUDA_KERNEL_LOOP(index, nthreads) {

    const int w = index % shuffled_width;
    const int h = (index / shuffled_width) % shuffled_height;
    const int c = (index / shuffled_width / shuffled_height) % shuffled_channels;
    const int n = index / shuffled_width / shuffled_height / shuffled_channels;
    int h0 = h / sigma;
    int w0 = w / sigma;
    int c0 = (h % sigma * sigma + w % sigma) * shuffled_channels + c;
    int offset = ((n * bottom_c + c0) * bottom_h + h0) * bottom_w + w0;

    top_data[index] = bottom_data[offset];
  }
}

template <typename Dtype>
void PixelShuffleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const int count = top[0]->count();
  Dtype* top_data = top[0]->mutable_gpu_data();
  PixelShuffleForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, bottom_data, top_data, sigma_,
      h_, w_, c_, shuffled_c, shuffled_h, shuffled_w);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void PixelShuffleBackward(const int nthreads, Dtype* bottom_diff, const Dtype* top_diff, int sigma,
                                    int bottom_h, int bottom_w, int bottom_c, int shuffled_channels, int shuffled_height, int shuffled_width) {
  CUDA_KERNEL_LOOP(index, nthreads) {

    const int w = index % shuffled_width;
    const int h = (index / shuffled_width) % shuffled_height;
    const int c = (index / shuffled_width / shuffled_height) % shuffled_channels;
    const int n = index / shuffled_width / shuffled_height / shuffled_channels;
    int h0 = h / sigma;
    int w0 = w / sigma;
    int c0 = (h % sigma * sigma + w % sigma) * shuffled_channels + c;
    int offset = ((n * bottom_c + c0) * bottom_h + h0) * bottom_w + w0;

    bottom_diff[offset] = top_diff[index];
  }
}

template <typename Dtype>
void PixelShuffleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  if (propagate_down[0]) {
    const int count = top[0]->count();
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    PixelShuffleBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, bottom_diff, top_diff, sigma_,
        h_, w_, c_, shuffled_c, shuffled_h, shuffled_w);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(PixelShuffleLayer);

}  // namespace caffe
